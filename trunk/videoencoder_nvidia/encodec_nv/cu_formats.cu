#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>


void checkCUDAError(const char *msg)                                                                                                                                   
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err ){
        fprintf(stderr, "ERROR[CUDA]:%s{%s}.\n", msg, hipGetErrorString( err ) );
        exit(EXIT_FAILURE);                                
    }
}



//Copy RGB data from shared memory region..
void copy_shmrgb_to_device(unsigned char*rgbs,
        unsigned char*devmem,//already allocated throuth hipMalloc()
        int rgbleft,int rgbtop,
        int rgbwidth,int rgbheight,
        int width,int height)
{

    int offset=(rgbtop*width)<<2;
    int offset_left=rgbleft<<2;
    
    int line_siz=width<<2;

    int h=0;

    for(h=rgbtop;h<rgbheight+rgbtop;h++){
        hipMemcpy(devmem+offset+offset_left,rgbs+offset+offset_left,rgbwidth<<2,hipMemcpyHostToDevice);
        offset+=line_siz;
    }

}




//for TEST ONLY,
void copy_caprgb_to_device(unsigned char*rgbs,
        unsigned char*devmem,//already allocated throuth hipMalloc()
        int patch_left,int patch_top,
        int patch_width,int patch_height,
        int width,int height)
{

    int rgb_offset=0;

    int offset=(patch_top*width)<<2;
    int offset_left=patch_left<<2;
    
    int line_siz=width<<2;

    int h;

    for(h=0;h<patch_height;h++){
        hipMemcpy(devmem+offset+offset_left,rgbs+rgb_offset,patch_width<<2,hipMemcpyHostToDevice);
        offset+=line_siz;
        rgb_offset+=(patch_width<<2);
    }

}







__global__ void 
convert_line_rgb_to_nv12(unsigned char*devrgb,int rgbstride,/*device mem*/
        unsigned char*oyuv,int ostride,int ovstride,/*device mem*/
        int width,int left,int top)
{
    int curline=threadIdx.x;

    unsigned char*rgb_p=devrgb+(curline+top)*rgbstride*4;
    unsigned char*luma_p=oyuv+(curline+top)*ostride;
    unsigned char*chroma_p=oyuv+(ovstride*ostride)+((curline+top)>>1)*ostride;

    int r,g,b;
    int y,u,v;

    int j;
    if(curline%2==0){
    //even line
        for(j=left;j<width+left;j++){

            b=*(rgb_p+j*4);
            g=*(rgb_p+j*4+1);
            r=*(rgb_p+j*4+2);

            y= 0.299*r + 0.587*g + 0.114*b;
            *(luma_p+j)=(char)y&0xff;

            if(j%2==0){
                u= -0.169*r - 0.331*g + 0.5*b+128;
                *(chroma_p+j)=(char)u&0xff;
            }
            
        }

    }else{
    //odd line

         for(j=left;j<width+left;j++){

            b=*(rgb_p+j*4);
            g=*(rgb_p+j*4+1);
            r=*(rgb_p+j*4+2);

            y= 0.299*r + 0.587*g + 0.114*b;
            *(luma_p+j)=(char)y&0xff;
          
            if(j%2==0){
                v= 0.5*r - 0.419*g - 0.081*b+128;
                *(chroma_p+j+1)=(char)v&0xff;
            } 
        }

    }

}





//FIXME

__global__ void 
convert_line_yv12_to_nv12(unsigned char*pdev,int istride,
        unsigned char*oyuv,int ostride,
        int width,int height)
{
    int curline=threadIdx.x;

    int yv12_luma_siz = istride*height;
    int yv12_chrome_siz = yv12_luma_siz>>2;

    int curpos=curline*istride;


    unsigned char*yv12_luma_p=pdev+curpos;
    unsigned char*yv12_v_p=pdev+yv12_luma_siz+(curpos>>1);
    unsigned char*yv12_u_p=pdev+yv12_luma_siz+yv12_chrome_siz+(curpos>>1);

    curpos=curline*ostride;

    unsigned char*nv12_luma_p=oyuv+curpos;
    unsigned char*nv12_chrome_p=oyuv+(height*ostride)+(curpos>>1);


    char val;

    int j;
    for(j=0;j<width;j++){

        val=*(yv12_luma_p+j);
        *(nv12_luma_p+j)=val;

        val=*(yv12_u_p+j);
        *(nv12_chrome_p)=val;

        val=*(yv12_v_p+j);
        *(nv12_chrome_p+1)=val;
    }

}












extern "C" void load_rgb_bgrx_cuda(
        unsigned char* oyuv,/*device*/
        unsigned char* devrgb,/*device */
        unsigned char*rgb, /*input data host*/
        int left,int top,int width,int height,//rgb patch rect
        int rgbwidth,int rgbheight,//rgb data size
        int ostride //yuv data height<pixel>
        )
{

    //Copy date from shared Memory to Device;
#if 0
    // Read rects from shm region.
    copy_shmrgb_to_device((unsigned char*)rgb,
        (unsigned char*)devrgb,//already allocated throuth hipMalloc()
        left,top,
        width,height,
        rgbwidth,rgbheight);

#else
    //for TEST :read rects from capture file.
    copy_caprgb_to_device((unsigned char*)rgb,
        (unsigned char*)devrgb,//already allocated throuth hipMalloc()
        left,top,
        width,height,
        rgbwidth,rgbheight);

#endif


    int ovstride=rgbheight;

//    fprintf(stderr,"rgbwidth:%d ostride:%d ovstride:%d, width:%d, left:%d, top:%d\n",rgbwidth,ostride,ovstride,width,left,top);
    convert_line_rgb_to_nv12<<<1,height>>>(devrgb,rgbwidth,
                                                oyuv,ostride,ovstride,
                                                width,left,top);

    hipDeviceSynchronize();

    checkCUDAError("Convert BGRA to NV12\n");

}






extern "C" void load_yuv_yv12_cuda(
        unsigned char* oyuv,/*device*/
        unsigned char* devyv12,/*device */
        unsigned char*iyuv, /*input data host*/
        int width,int height,/*real size*/
        int istride,int ostride
        )
{



//  Load yv12 to device buffer
//TODO

    int in_luma_siz=istride*height;
    int out_luma_siz=ostride*height;
    int in_chroma_siz=in_luma_siz>>2;
    int out_chroma_siz=out_luma_siz>>2;

    unsigned char*in_luma_p=iyuv;
    unsigned char*out_luma_p=devyv12;


    unsigned char*in_v_p=iyuv+in_luma_siz;
    unsigned char*out_v_p=devyv12+out_luma_siz;

    unsigned char*in_u_p=iyuv+in_luma_siz+in_chroma_siz;
    unsigned char*out_u_p=devyv12+out_luma_siz+out_chroma_siz;


    int j;

    for(j=0;j<height;j++){
        //y
        memcpy(out_luma_p+j*ostride,in_luma_p+j*istride,width);
    }

    for(j=0;j<(height>>1);j++){
        //v
        memcpy(out_v_p+((j*ostride)>>1),in_v_p+((j*istride)>>1),width>>1);
        //u
        memcpy(out_u_p+((j*ostride)>>1),in_u_p+((j*istride)>>1),width>>1);
    }




//    fprintf(stderr,"rgbwidth:%d ostride:%d ovstride:%d, width:%d, left:%d, top:%d\n",rgbwidth,ostride,ovstride,width,left,top);
    convert_line_yv12_to_nv12<<<1,height>>>(devyv12,istride,
                                                oyuv,ostride,
                                                width,height);

    hipDeviceSynchronize();

    checkCUDAError("Convert YV12 to NV12\n");

}










