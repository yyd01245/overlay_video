#include "hip/hip_runtime.h"

#include<stdio.h>
#include<stdlib.h>
#include<string.h>

//#include"formats.h"


void checkCUDAError(const char *msg)                                                                                                                                   
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err ){
        fprintf(stderr, "ERROR[CUDA]:%s{%s}.\n", msg, hipGetErrorString( err ) );
        exit(EXIT_FAILURE);                                
    }
}



//Copy RGB data from shared memory region..
inline void copy_shmrgb_to_device(unsigned char*rgbs,
        unsigned char*devmem,//already allocated throuth hipMalloc()
        int rgbleft,int rgbtop,
        int rgbwidth,int rgbheight,
        int width,int height)
{

    int offset=(rgbtop*width)<<2;
    int offset_left=rgbleft<<2;
    
    int line_siz=width<<2;

    int h=0;

    for(h=rgbtop;h<rgbheight+rgbtop;h++){
        hipMemcpy(devmem+offset+offset_left,rgbs+offset+offset_left,rgbwidth<<2,hipMemcpyHostToDevice);
        offset+=line_siz;
    }

}




//for TEST ONLY,
inline void copy_caprgb_to_device(unsigned char*rgbs,
        unsigned char*devmem,//already allocated throuth hipMalloc()
        int patch_left,int patch_top,
        int patch_width,int patch_height,
        int width,int height)
{

    int rgb_offset=0;

    int offset=(patch_top*width)<<2;
    int offset_left=patch_left<<2;
    
    int line_siz=width<<2;

    int h;

    for(h=0;h<patch_height;h++){
        hipMemcpy(devmem+offset+offset_left,rgbs+rgb_offset,patch_width<<2,hipMemcpyHostToDevice);
        offset+=line_siz;
        rgb_offset+=(patch_width<<2);
    }

}







__global__ void 
convert_line_rgb_to_nv12(unsigned char*devrgb,int rgbstride,/*device mem*/
        unsigned char*oyuv,int ostride,int ovstride,/*device mem*/
        int width,int left,int top)
{
    int curline=threadIdx.x;

    unsigned char*rgb_p=devrgb+(curline+top)*rgbstride*4;
    unsigned char*luma_p=oyuv+(curline+top)*ostride;
    unsigned char*chroma_p=oyuv+(ovstride*ostride)+((curline+top)>>1)*ostride;

    int r,g,b;
    int y,u,v;

    int j;
    if(curline%2==0){
    //even line
        for(j=left;j<width+left;j++){

            b=*(rgb_p+j*4);
            g=*(rgb_p+j*4+1);
            r=*(rgb_p+j*4+2);

            y= 0.299*r + 0.587*g + 0.114*b;
            *(luma_p+j)=(char)y&0xff;

            if(j%2==0){
                u= -0.169*r - 0.331*g + 0.5*b+128;
                *(chroma_p+j)=(char)u&0xff;
            }
            
        }

    }else{
    //odd line

         for(j=left;j<width+left;j++){

            b=*(rgb_p+j*4);
            g=*(rgb_p+j*4+1);
            r=*(rgb_p+j*4+2);

            y= 0.299*r + 0.587*g + 0.114*b;
            *(luma_p+j)=(char)y&0xff;
          
            if(j%2==0){
                v= 0.5*r - 0.419*g - 0.081*b+128;
                *(chroma_p+j+1)=(char)v&0xff;
            } 
        }

    }

}





//FIXME

__global__ void 
convert_line_yv12_to_nv12(unsigned char*pdev,int istride,
        unsigned char*oyuv,int ostride,
        int width,int height)
{
    int curline=threadIdx.x;

    int yv12_luma_siz = istride*height;
    int yv12_chroma_siz = yv12_luma_siz>>2;

    int curpos=curline*istride;


    unsigned char*yv12_luma_p=pdev+curpos;
    unsigned char*yv12_v_p=pdev+yv12_luma_siz+(curpos>>1);
    unsigned char*yv12_u_p=pdev+yv12_luma_siz+yv12_chroma_siz+(curpos>>1);

    curpos=curline*ostride;

    unsigned char*nv12_luma_p=oyuv+curpos;
    unsigned char*nv12_chroma_p=oyuv+(height*ostride)+(curpos>>1);


    char val;

    int j;
    for(j=0;j<width;j++){

        val=*(yv12_luma_p+j);
        *(nv12_luma_p+j)=val;

        val=*(yv12_u_p+j);
        *(nv12_chroma_p)=val;

        val=*(yv12_v_p+j);
        *(nv12_chroma_p+1)=val;
    }

}












extern "C" void load_rgb_bgrx_cuda(
        unsigned char* oyuv,/*device*/
        unsigned char* devrgb,/*device */
        unsigned char*rgb, /*input data host*/
        int left,int top,int width,int height,//rgb patch rect
        int rgbwidth,int rgbheight,//rgb data size
        int ostride //yuv data height<pixel>
        )
{

    //Copy date from shared Memory to Device;
#if 1
    // Read rects from shm region.
    copy_shmrgb_to_device((unsigned char*)rgb,
        (unsigned char*)devrgb,//already allocated throuth hipMalloc()
        left,top,
        width,height,
        rgbwidth,rgbheight);

#else
    //for TEST :read rects from capture file.
    copy_caprgb_to_device((unsigned char*)rgb,
        (unsigned char*)devrgb,//already allocated throuth hipMalloc()
        left,top,
        width,height,
        rgbwidth,rgbheight);

#endif


    int ovstride=rgbheight;

//    fprintf(stderr,"rgbwidth:%d ostride:%d ovstride:%d, width:%d, left:%d, top:%d\n",rgbwidth,ostride,ovstride,width,left,top);
    convert_line_rgb_to_nv12<<<1,height>>>(devrgb,rgbwidth,
                                                oyuv,ostride,ovstride,
                                                width,left,top);

    hipDeviceSynchronize();

    checkCUDAError("Convert BGRA to NV12\n");

}






extern "C" void load_yuv_yv12_cuda(
        unsigned char* oyuv,/*device*/
        unsigned char* devyv12,/*device */
        unsigned char*iyuv, /*input data host*/
        int width,int height,/*real size*/
        int istride,int ostride
        )
{



//  Load yv12 to device buffer
//TODO

    int in_luma_siz=istride*height;
    int out_luma_siz=ostride*height;
    int in_chroma_siz=in_luma_siz>>2;
    int out_chroma_siz=out_luma_siz>>2;

    unsigned char*in_luma_p=iyuv;
    unsigned char*out_luma_p=devyv12;


    unsigned char*in_v_p=iyuv+in_luma_siz;
    unsigned char*out_v_p=devyv12+out_luma_siz;

    unsigned char*in_u_p=iyuv+in_luma_siz+in_chroma_siz;
    unsigned char*out_u_p=devyv12+out_luma_siz+out_chroma_siz;


    int j;

    for(j=0;j<height;j++){
        //y
        memcpy(out_luma_p+j*ostride,in_luma_p+j*istride,width);
    }

    for(j=0;j<(height>>1);j++){
        //v
        memcpy(out_v_p+((j*ostride)>>1),in_v_p+((j*istride)>>1),width>>1);
        //u
        memcpy(out_u_p+((j*ostride)>>1),in_u_p+((j*istride)>>1),width>>1);
    }




//    fprintf(stderr,"rgbwidth:%d ostride:%d ovstride:%d, width:%d, left:%d, top:%d\n",rgbwidth,ostride,ovstride,width,left,top);
    convert_line_yv12_to_nv12<<<1,height>>>(devyv12,istride,
                                                oyuv,ostride,
                                                width,height);

    hipDeviceSynchronize();

    checkCUDAError("Convert YV12 to NV12\n");

}


/***************************************************/
/***************************************************/
/***************************************************/
/***************************************************/



extern"C"{


inline void rgb2yuv_pixel(
        unsigned char r,
        unsigned char g,
        unsigned char b,
        unsigned char*y,
        unsigned char*u,
        unsigned char*v
        ){
#if 0
        //YCbCr
            *y=(0.257*r)+(0.504*g)+(0.098*b)+16;
            *u=-(0.148 * r) - (0.291 * g) + (0.439 * b) + 128;
            *v=(0.439*r)-(0.368*g)+(0.071*b)+128;
#else
        //YUV Intel IPP的BT.709
            *y= 0.299*r + 0.587*g + 0.114*b;
            *u= -0.169*r - 0.331*g + 0.5*b+128;
            *v= 0.5*r - 0.419*g - 0.081*b+128;
#endif

}




/*For Test*/
void load_rgb_bgrx_(unsigned char*yuv,unsigned char*rgb,
        int left,int top,int width,int height,//patch rectangle
        int rgbheight,
        int ostride)

{
    //assert left top width height are even;
    //
    int luma_off=ostride*rgbheight;

    unsigned char*luma_p;
    unsigned char*chroma_p;
    unsigned char*rgb_p;
    

    int r,g,b;
    int y,u,v;
//    fprintf(stderr,"LOAD {x:%d, y:%d, w:%d, h:%d, ww:%d, hh:%d }\n",left,top,width,height,stride,vstride);

    int i,j;
    for(i=top;i<height+top;i++){
        //rows
        rgb_p=rgb+width*(i-top)*4;
        luma_p=yuv+ostride*i;
        chroma_p=yuv+luma_off+ostride*(i/2);

        for(j=left;j<width+left;j++){

            b=*(rgb_p+(j-left)*4);
            g=*(rgb_p+(j-left)*4+1);
            r=*(rgb_p+(j-left)*4+2);

            y= 0.299*r + 0.587*g + 0.114*b;
            *(luma_p+j)=(char)y&0xff;

            if(i%2==0 && j%2==0){
                u= -0.169*r - 0.331*g + 0.5*b+128;
                *(chroma_p+j)=(char)u&0xff;
            }
            
            if(i%2==1 && j%2==0){
                v= 0.5*r - 0.419*g - 0.081*b+128;
                *(chroma_p+j+1)=(char)v&0xff;
            } 
        }
    }
}





void load_rgb_bgrx_2(unsigned char*yuv,unsigned char*rgb,
        int left,int top,int width,int height,//patch rectangle
        int rgbheight,
        int ostride)

{
    //assert left top width height are even;
    //
    int luma_off=ostride*rgbheight;

    unsigned char*luma_p0,*luma_p1;
    unsigned char*chroma_p;
    unsigned char*rgb_p0,*rgb_p1;
   
    int au;//(u1+u2+u3+u4)/4
    int av;//

    unsigned char r,g,b;
    unsigned char y,u,v;
//    fprintf(stderr,"LOAD {x:%d, y:%d, w:%d, h:%d }\n",left,top,width,height);//,stride,vstride);

    int i,j;
    for(i=top;i<height+top;i+=2){
        //rows
        rgb_p0=rgb+width*(i-top)*4;
        rgb_p1=rgb+width*(i-top+1)*4;
        luma_p0=yuv+ostride*i;
        luma_p1=yuv+ostride*(i+1);
        chroma_p=yuv+luma_off+ostride*(i/2);

        for(j=left;j<width+left;j++){

            b=*(rgb_p0+(j-left)*4);
            g=*(rgb_p0+(j-left)*4+1);
            r=*(rgb_p0+(j-left)*4+2);

            rgb2yuv_pixel(r,g,b,&y,&u,&v);
            *(luma_p0+j)=(char)y&0xff;
            au+=u;
            av+=v;
///////////
            b=*(rgb_p1+(j-left)*4);
            g=*(rgb_p1+(j-left)*4+1);
            r=*(rgb_p1+(j-left)*4+2);

            rgb2yuv_pixel(r,g,b,&y,&u,&v);
            *(luma_p1+j)=(char)y&0xff;
            au+=u;
            av+=v;

            if(j%2==0){
                *(chroma_p+j)=(au>>2)&0xff;
                *(chroma_p+j+1)=(av>>2)&0xff;
                av=au=0;
            }

        }
    }
}


/*
void load_rgb_bgrx(unsigned char*yuv,unsigned char*rgb,
        int left,int top,int width,int height,//patch rectangle
        int rgbheight,
        int ostride)
*/

void load_rgb_bgrx(
        unsigned char*bgrx,
        unsigned char*nv12, 
        int pleft,int ptop,int pwidth,int pheight,//rgb patch rect
        int width,int height,//rgb data size
        int sstride,
        int dstride //yuv data stride<pixel>
        )

{
    //assert left top width height are even;
    //

    if (sstride == 0)
        sstride = width;

    if (dstride == 0)
        dstride = width;

    int luma_off=dstride*height;

    unsigned char*luma_p0,*luma_p1;
    unsigned char*chroma_p;
    unsigned char*rgb_p0,*rgb_p1;
   
    int au;//(u1+u2+u3+u4)/4
    int av;//

    unsigned char r,g,b;
    unsigned char y,u,v;
//    fprintf(stderr,"LOAD {x:%d, y:%d, w:%d, h:%d, ww:%d, hh:%d }\n",left,top,width,height,stride,vstride);

    int i,j;
    for(i=ptop;i<pheight+ptop;i+=2){
        //rows
        rgb_p0=bgrx+sstride*(i)*4;
        rgb_p1=bgrx+sstride*(i+1)*4;
        luma_p0=nv12+dstride*i;
        luma_p1=nv12+dstride*(i+1);
        chroma_p=nv12+luma_off+dstride*(i/2);

        for(j=pleft;j<pwidth+pleft;j++){

            b=*(rgb_p0+j*4);
            g=*(rgb_p0+j*4+1);
            r=*(rgb_p0+j*4+2);

            rgb2yuv_pixel(r,g,b,&y,&u,&v);
            *(luma_p0+j)=(char)y&0xff;
            au=u;
//            av=v;
///////////
            b=*(rgb_p1+j*4);
            g=*(rgb_p1+j*4+1);
            r=*(rgb_p1+j*4+2);

            rgb2yuv_pixel(r,g,b,&y,&u,&v);
            *(luma_p1+j)=(char)y&0xff;
//            au+=u;
            av=v;

            if(j%2==0){
                *(chroma_p+j)=au&0xff;
                *(chroma_p+j+1)=av&0xff;
//                av=au=0;
            }
        }
    }
}




#if 0
void load_rgb_bgrx__(
        unsigned char*bgrx,
        unsigned char*nv12, 
        int pleft,int ptop,int pwidth,int pheight,//rgb patch rect
        int width,int height,//rgb data size
        int sstride,
        int dstride //yuv data stride<pixel>
        )
{
   
    unsigned char*luma_p=nv12;
    unsigned char*chroma_p;

    unsigned char*rgb_p=bgrx;

    if (sstride == 0)
        sstride = width;

    if (dstride == 0)
        dstride = width;

    chroma_p=luma_p+dstride*height;

    unsigned char b,g,r;
    unsigned char y,u,v;

    int i,j;

    for(i=ptop;i<pheight;i+=2){//vertical
//==============
    rgb_p=bgrx+i*sstride*4;
    luma_p=nv12+dstride*i;
    chroma_p=nv12+dstride+height+dstride*(i/2);
        for(j=pleft;j<pwidth+pleft;j++){

            b=*(rgb_p+j*4);
            g=*(rgb_p+j*4+1);
            r=*(rgb_p+j*4+2);

            y= 0.299*r + 0.587*g + 0.114*b;
            *(luma_p+j)=(char)y&0xff;

//            if(j%2==0){
                u= -0.169*r - 0.331*g + 0.5*b+128;
                *(chroma_p+j)=(char)u&0xff;
//            }
        }
    //odd line
        rgb_p+=sstride*4;
        luma_p+=dstride;
         for(j=pleft;j<pwidth+pleft;j++){

            b=*(rgb_p+j*4);
            g=*(rgb_p+j*4+1);
            r=*(rgb_p+j*4+2);

            y= 0.299*r + 0.587*g + 0.114*b;
            *(luma_p+j)=(char)y&0xff;
          
//            if(j%2==0){
                v= 0.5*r - 0.419*g - 0.081*b+128;
                *(chroma_p+j+1)=(char)v&0xff;
//            } 
        }

//    }


    }





}


#endif



void load_yuv_yv12(unsigned char*yv12,unsigned char*nv12,int width,int height,int sstride,int dstride)
{

    unsigned char*nv12_luma=nv12;
    unsigned char*nv12_chroma;

    unsigned char*yv12_luma=yv12;
    unsigned char*yv12_v;
    unsigned char*yv12_u;



    if (sstride == 0)
        sstride = width;

    if (dstride == 0)
        dstride = width;

    nv12_chroma=nv12_luma+dstride*height;
    
    yv12_v=yv12_luma+sstride*height;
    yv12_u=yv12_v+sstride*height/4;


    int y;
    int x;

    for (y = 0 ; y < height ; y++){
        memcpy(nv12_luma + (dstride*y), yv12_luma + (sstride*y) , width);
    }

    for (y = 0 ; y < height/2 ; y++){
        for (x= 0 ; x < width; x=x+2){
            nv12_chroma[(y*dstride) + x] =    yv12_v[((sstride/2)*y) + (x >>1)];
            nv12_chroma[(y*dstride) +(x+1)] = yv12_u[((sstride/2)*y) + (x >>1)];
        }
    }
   

}


void load_yuv_nv12(unsigned char*inyuv, unsigned char*outyuv,int width,int height,int istride,int ostride)
{

    if(istride==0)
        istride=width;
    if(ostride==0)
        ostride=width;


    unsigned char*inyuv_chroma=inyuv+width*istride;
    unsigned char*outyuv_chroma=outyuv+width*ostride;

    int y;

    for(y=0;y<height;y++){
        memcpy(outyuv+y*ostride,inyuv+y*istride,width);
    }

    for(y=0;y<height/2;y++){
        memcpy(outyuv_chroma+y*ostride/2,inyuv_chroma+y*istride/2,width/2);
    }

}



}//extern "C"





